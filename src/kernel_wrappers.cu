#include <kernel_wrappers.h>
#include <kmeans_cuda.h>

void call_compute_assignments_kernel(float *points, float *centroids, uint32_t *assignments, uint32_t n_points, uint32_t n_centroids, uint32_t n_dims) {
    float *d_points, *d_centroids, *d_accumulator;
    uint32_t *d_assignments, *d_sizes, *d_n_points, *d_n_centroids, *d_n_dims;

    int threads_per_block = 16;
    int blocks = n_points / (threads_per_block * PTS_PER_THREAD);
    size_t shmem_size = (threads_per_block * PTS_PER_THREAD * n_dims + SHM_K * SHM_DIM) * sizeof(float); 

    host_to_device_init_transfer(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );

    compute_assignments_kernel<<< blocks, threads_per_block, shmem_size >>> (d_points, d_centroids, d_assignments, d_n_points, d_n_centroids, d_n_dims);

    hipDeviceSynchronize();

    device_to_host_transfer_free(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );
}