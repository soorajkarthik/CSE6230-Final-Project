#include "hip/hip_runtime.h"
#include <kmeans_cuda.h>
#include <dataset.h>
#include <cstdio>

void host_to_device_init_transfer(
    float *points, float **d_points, 
    float *centroids, float **d_centroids,
    uint32_t *assignments, uint32_t **d_assignments,
    float **d_accumulator, uint32_t **d_sizes,
    uint32_t n_points, uint32_t **d_n_points,
    uint32_t n_centroids, uint32_t **d_n_centroids,
    uint32_t n_dims, uint32_t **d_n_dims) {

    hipMalloc(d_points,      n_points * n_dims * sizeof(float));
    hipMalloc(d_centroids,   n_centroids * n_dims * sizeof(float));
    hipMalloc(d_assignments, n_points * sizeof(uint32_t));
    hipMalloc(d_accumulator, n_centroids * n_dims * NUM_PRIV_COPIES * sizeof(float));
    hipMalloc(d_sizes,       n_centroids * sizeof(uint32_t));
    hipMalloc(d_n_points,    sizeof(uint32_t));
    hipMalloc(d_n_centroids, sizeof(uint32_t));
    hipMalloc(d_n_dims,      sizeof(uint32_t));

    hipMemcpy(*d_points,      points,       n_points * n_dims * sizeof(float),    hipMemcpyHostToDevice);
    hipMemcpy(*d_centroids,   centroids,    n_centroids * n_dims * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_n_points,    &n_points,    sizeof(uint32_t),                     hipMemcpyHostToDevice);
    hipMemcpy(*d_n_centroids, &n_centroids, sizeof(uint32_t),                     hipMemcpyHostToDevice);
    hipMemcpy(*d_n_dims,      &n_dims, 	 sizeof(uint32_t),                     hipMemcpyHostToDevice);

    hipMemset(d_accumulator, 0, n_centroids * n_dims * NUM_PRIV_COPIES * sizeof(float));
}

void device_to_host_transfer_free(
    float *points, float **d_points, 
    float *centroids, float **d_centroids,
    uint32_t *assignments, uint32_t **d_assignments,
    float **d_accumulator, uint32_t **d_sizes,
    uint32_t n_points, uint32_t **d_n_points,
    uint32_t n_centroids, uint32_t **d_n_centroids,
    uint32_t n_dims, uint32_t **d_n_dims) {

    hipMemcpy(centroids,   *d_centroids, n_centroids * n_dims * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(assignments, *d_assignments, n_points * sizeof(uint32_t),        hipMemcpyDeviceToHost);

    hipFree(&d_points);
    hipFree(&d_centroids);
    hipFree(&d_assignments);
    hipFree(&d_accumulator);
    hipFree(&d_sizes);
    hipFree(&d_n_points);
    hipFree(&d_n_centroids);
    hipFree(&d_n_dims);
}

__global__ void compute_assignments_kernel(
    float *__restrict__ points, 
    float *__restrict__ centroids, 
    uint32_t *__restrict__ assignments, 
    uint32_t *__restrict__ n_points, 
    uint32_t *__restrict__ n_centroids, 
    uint32_t *__restrict__ n_dims) {

    uint32_t K = *n_centroids, D = *n_dims;

    extern __shared__ float shmem[];

    float *shm_points = shmem;
    float *shm_centroids = shm_points + PTS_PER_THREAD * D * blockDim.x; 
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    int point_idx = tid * PTS_PER_THREAD;
    int point_offset = point_idx * D;

    int shm_point_idx = threadIdx.x * PTS_PER_THREAD;
    int shm_point_offset = shm_point_idx * D;

    float dists[PTS_PER_THREAD][SHM_K];
    float min_dists[PTS_PER_THREAD];
    uint32_t local_assignments[PTS_PER_THREAD];

    // Load points into shared memory
    #pragma unroll
    for(int i = 0; i < PTS_PER_THREAD; i++) {
        for(int d = 0; d < D; d++) {
            shm_points[shm_point_offset + i * D + d] = points[point_offset + i * D + d];
        }
        min_dists[i] = 1e30;
    }

    // Tiled loop over K
    for(int k_block = 0; k_block < K; k_block += SHM_K) {

        // Clear distances
        #pragma unroll
        for(int i = 0; i < PTS_PER_THREAD; i++) {

            #pragma unroll
            for(int j = 0; j < SHM_K; j++) {
                dists[i][j] = 0;
            }
        }

        // Tiled loop over D
        for(int d_block = 0; d_block < D; d_block += SHM_DIM) {

            // Load centroids into shared memory
            int k, d, p;
            for(int shm_idx = threadIdx.x; shm_idx < SHM_K * SHM_DIM; shm_idx += blockDim.x) {
                
                k = shm_idx / SHM_DIM;
                d = shm_idx % SHM_DIM;


                shm_centroids[shm_idx] = centroids[(k + k_block) * D + (d + d_block)];
            }

            __syncthreads();

            // Accumulate distances for this set of dimensions
            #pragma unroll
            for(k = 0; k < SHM_K; k++) {

                #pragma unroll
                for(d = 0; d < SHM_DIM; d++) {

                    float centroid_val = shm_centroids[k * SHM_DIM + d];

                    #pragma unroll
                    for(p = 0; p < PTS_PER_THREAD; p++) {
                        float val = centroid_val - shm_points[shm_point_offset + p * D + (d + d_block)];
                        dists[p][k] += val * val;
                    }
                }
            }

            __syncthreads();
        }


        // Reassign
        #pragma unroll
        for(int k = 0; k < SHM_K; k++) {

            #pragma unroll
            for(int p = 0; p < PTS_PER_THREAD; p++) {

                if(min_dists[p] > dists[p][k]) {
                    min_dists[p] = dists[p][k];
                    local_assignments[p] = k + k_block;
                }
            }
        }
    }

    // Write final assignments to global memory
    #pragma unroll
    for(int p = 0; p < PTS_PER_THREAD; p++) {
        assignments[point_idx + p] = local_assignments[p];
    }
}

__device__ float* get_privatized_pointer(
    float *ptr, 
    uint32_t n_vecs, 
    uint32_t n_dims) {
    
    float *res = ptr;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int wid = tid / warpSize;
    
    // All processors in the same warp assigned to the same copy
    // since they write to consecutive elements
    res += (n_vecs * n_dims) * (wid % NUM_PRIV_COPIES);

    return res;
} 

__global__ void reduce_private_copies_kernel(
    float *result, 
    uint32_t *n_centroids, 
    uint32_t *n_dims) {

    uint32_t K = *n_centroids, D = *n_dims;

    int size = K * D;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_threads = blockDim.x * gridDim.x;

    for(int i = tid; i < size; i += n_threads) {

        float accumulator = result[i];
        float *copy_ptr = result;

        #pragma unroll
        for(int copy = 1; copy < NUM_PRIV_COPIES; copy++) {
            copy_ptr += size;
            accumulator += copy_ptr[i];
        }

        result[i] = accumulator;
    }
}

__global__ void divide_centroids_kernel(
    float *centroids, 
    uint32_t *counts, 
    uint32_t *n_centroids, 
    uint32_t *n_dims) {

    uint32_t K = *n_centroids, D = *n_dims;

    int size = K * D;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_threads = blockDim.x * gridDim.x;

    for(int i = tid; i < size; i += n_threads) {
        int k = i / D;
        float div = counts[k];

        if(div > 0) {
            centroids[i] /= div;
        }
    }
}

__global__ void accumulate_cluster_members_kernel(
    float *points, 
    float *accumulator, 
    uint32_t *assignments, 
    uint32_t *counts, 
    uint32_t *n_points,
    uint32_t *n_centroids,
    uint32_t *n_dims) {

    uint32_t N = *n_points, K = *n_centroids, D = *n_dims;

    float *priv_accumulator = get_privatized_pointer(accumulator, K, D);
    
    int size = N * D;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_threads = blockDim.x * gridDim.x;

    for(int i = tid; i < size; i += n_threads) {
        
        int idx = i / D;
        int dim = i % D;

        float val = points[i];
        int cluster = assignments[idx];
        float *acc_val_ptr = &priv_accumulator[cluster * D + dim];

        atomicAdd(acc_val_ptr, val);
        if(dim == 0) {
            atomicAdd(&counts[cluster], 1);
        }
    }
}

__global__ void fused_assignment_accumulate_kernel(    
    float *points, 
    float *centroids, 
    float *accumulator, 
    uint32_t *assignments, 
    uint32_t *counts,
    uint32_t *n_points,
    uint32_t *n_centroids,
    uint32_t *n_dims) {

    uint32_t K = *n_centroids, D = *n_dims;

    extern __shared__ float shmem[];

    float *shm_points = shmem;
    float *shm_centroids = shm_points + PTS_PER_THREAD * D * blockDim.x; 
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    int point_idx = tid * PTS_PER_THREAD;
    int point_offset = point_idx * D;

    int shm_point_idx = threadIdx.x * PTS_PER_THREAD;
    int shm_point_offset = shm_point_idx * D;

    float dists[PTS_PER_THREAD][SHM_K];
    float min_dists[PTS_PER_THREAD];
    uint32_t local_assignments[PTS_PER_THREAD];

    // Load points into shared memory
    #pragma unroll
    for(int i = 0; i < PTS_PER_THREAD; i++) {
        for(int d = 0; d < D; d++) {
            shm_points[shm_point_offset + i * D + d] = points[point_offset + i * D + d];
        }
        min_dists[i] = 1e30;
    }

    // Tiled loop over K
    for(int k_block = 0; k_block < K; k_block += SHM_K) {

        // Clear distances
        #pragma unroll
        for(int i = 0; i < PTS_PER_THREAD; i++) {

            #pragma unroll
            for(int j = 0; j < SHM_K; j++) {
                dists[i][j] = 0;
            }
        }

        // Tiled loop over D
        for(int d_block = 0; d_block < D; d_block += SHM_DIM) {

            // Load centroids into shared memory
            int k, d, p;
            for(int shm_idx = threadIdx.x; shm_idx < SHM_K * SHM_DIM; shm_idx += blockDim.x) {
                
                k = shm_idx / SHM_DIM;
                d = shm_idx % SHM_DIM;


                shm_centroids[shm_idx] = centroids[(k + k_block) * D + (d + d_block)];
            }

            __syncthreads();

            // Accumulate distances for this set of dimensions
            #pragma unroll
            for(k = 0; k < SHM_K; k++) {

                #pragma unroll
                for(d = 0; d < SHM_DIM; d++) {

                    float centroid_val = shm_centroids[k * SHM_DIM + d];

                    #pragma unroll
                    for(p = 0; p < PTS_PER_THREAD; p++) {
                        float val = centroid_val - shm_points[shm_point_offset + p * D + (d + d_block)];
                        dists[p][k] += val * val;
                    }
                }
            }

            __syncthreads();
        }


        // Reassign
        #pragma unroll
        for(int k = 0; k < SHM_K; k++) {

            #pragma unroll
            for(int p = 0; p < PTS_PER_THREAD; p++) {

                if(min_dists[p] > dists[p][k]) {
                    min_dists[p] = dists[p][k];
                    local_assignments[p] = k + k_block;
                }
            }
        }
    }

    // Write final assignments to global memory
    #pragma unroll
    for(int p = 0; p < PTS_PER_THREAD; p++) {
        uint32_t cluster = local_assignments[p];
        assignments[point_idx + p] = cluster;
        atomicAdd(&counts[cluster], 1);
    }

    // Accumulate 
    float *priv_accumulator = get_privatized_pointer(accumulator, K, D);
    for(uint32_t d = 0; d < D; d++) {

        #pragma unroll
        for(int p = 0; p < PTS_PER_THREAD; p++) {
            uint32_t cluster = local_assignments[p];
            float val = shm_points[shm_point_offset + p * D + d];
            float *acc_val_ptr = &priv_accumulator[cluster * D + d];

            atomicAdd(acc_val_ptr, val);
        }
    }
}


KMeansResult Dataset::kmeans_cuda(uint32_t n_centroids, uint32_t max_iters, float tol) {
    float *centroids = random_points(n_centroids);
    uint32_t *assignments = new uint32_t[n_points]; 

    vector<float> time_per_iter;

    int threads_per_block = 16;
    int blocks_assignment = n_points / (threads_per_block * PTS_PER_THREAD);
    
    int calcs_per_thread = 16;
    int blocks_accumulate = n_points * n_dims / (threads_per_block * calcs_per_thread);
    int blocks_reduce_divide = n_centroids * n_dims / (threads_per_block * calcs_per_thread);

    size_t shmem_size = (threads_per_block * PTS_PER_THREAD * n_dims + SHM_K * SHM_DIM) * sizeof(float); 

    float *d_points, *d_centroids, *d_accumulator;
    uint32_t *d_assignments, *d_sizes, *d_n_points, *d_n_centroids, *d_n_dims;
    
    host_to_device_init_transfer(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );

    compute_assignments_kernel<<< blocks_assignment, threads_per_block, shmem_size >>> (d_points, d_centroids, d_assignments, d_n_points, d_n_centroids, d_n_dims);
    accumulate_cluster_members_kernel<<< blocks_accumulate, threads_per_block >>> (d_points, d_accumulator, d_assignments, d_sizes, d_n_points, d_n_centroids, d_n_dims);

    hipDeviceSynchronize();

    device_to_host_transfer_free(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );
    
    return KMeansResult(centroids, n_centroids, assignments, time_per_iter);
}
