#include <dataset.h>
#include <kernels.h>
#include <device_utils.h>
#include <consts.h>

KMeansResult Dataset::kmeans_cuda(uint32_t n_centroids, uint32_t max_iters, float tol) {
    float *centroids = random_points(n_centroids);
    uint32_t *assignments = new uint32_t[n_points]; 

    vector<float> time_per_iter;

    int threads_per_block = 16;
    int blocks_assignment = n_points / (threads_per_block * PTS_PER_THREAD);
    
    int calcs_per_thread = 16;
    int blocks_accumulate = n_points * n_dims / (threads_per_block * calcs_per_thread);
    int blocks_reduce_divide = n_centroids * n_dims / (threads_per_block * calcs_per_thread);

    size_t shmem_size = (threads_per_block * PTS_PER_THREAD * n_dims + SHM_K * SHM_DIM) * sizeof(float); 

    float *d_points, *d_centroids, *d_accumulator;
    uint32_t *d_assignments, *d_sizes, *d_n_points, *d_n_centroids, *d_n_dims;
    
    host_to_device_init_transfer(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );

    compute_assignments_kernel<<< blocks_assignment, threads_per_block, shmem_size >>> (d_points, d_centroids, d_assignments, d_n_points, d_n_centroids, d_n_dims);
    accumulate_cluster_members_kernel<<< blocks_accumulate, threads_per_block >>> (d_points, d_accumulator, d_assignments, d_sizes, d_n_points, d_n_centroids, d_n_dims);

    hipDeviceSynchronize();

    device_to_host_transfer_free(
        points, &d_points,
        centroids, &d_centroids,
        assignments, &d_assignments,
        &d_accumulator, &d_sizes,
        n_points, &d_n_points,
        n_centroids, &d_n_centroids,
        n_dims, &d_n_dims
    );
    
    return KMeansResult(centroids, n_centroids, assignments, time_per_iter);
}
