#include <device_utils.h>
#include <kernels.h>
#include <consts.h>

void host_to_device_init_transfer(
    float *points, float **d_points, 
    float *centroids, float **d_centroids,
    uint32_t *assignments, uint32_t **d_assignments,
    float **d_accumulator, uint32_t **d_counts,
    uint32_t n_points, uint32_t **d_n_points,
    uint32_t n_centroids, uint32_t **d_n_centroids,
    uint32_t n_dims, uint32_t **d_n_dims) {

    hipMalloc(d_points,      n_points * n_dims * sizeof(float));
    hipMalloc(d_centroids,   n_centroids * n_dims * sizeof(float));
    hipMalloc(d_assignments, n_points * sizeof(uint32_t));
    hipMalloc(d_accumulator, n_centroids * n_dims * NUM_PRIV_COPIES * sizeof(float));
    hipMalloc(d_counts,       n_centroids * sizeof(uint32_t));
    hipMalloc(d_n_points,    sizeof(uint32_t));
    hipMalloc(d_n_centroids, sizeof(uint32_t));
    hipMalloc(d_n_dims,      sizeof(uint32_t));

    hipMemcpy(*d_points,      points,       n_points * n_dims * sizeof(float),    hipMemcpyHostToDevice);
    hipMemcpy(*d_centroids,   centroids,    n_centroids * n_dims * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_n_points,    &n_points,    sizeof(uint32_t),                     hipMemcpyHostToDevice);
    hipMemcpy(*d_n_centroids, &n_centroids, sizeof(uint32_t),                     hipMemcpyHostToDevice);
    hipMemcpy(*d_n_dims,      &n_dims, 	 sizeof(uint32_t),                     hipMemcpyHostToDevice);

    hipMemset(*d_assignments, 0, n_points * sizeof(uint32_t));
    hipMemset(*d_accumulator, 0, n_centroids * n_dims * NUM_PRIV_COPIES * sizeof(float));
    hipMemset(*d_counts,      0, n_centroids * sizeof(uint32_t));
}

void device_to_host_transfer_free(
    float *points, float **d_points, 
    float *centroids, float **d_centroids,
    uint32_t *assignments, uint32_t **d_assignments,
    float **d_accumulator, uint32_t **d_counts,
    uint32_t n_points, uint32_t **d_n_points,
    uint32_t n_centroids, uint32_t **d_n_centroids,
    uint32_t n_dims, uint32_t **d_n_dims) {

    hipMemcpy(centroids,   *d_centroids,   n_centroids * n_dims * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(assignments, *d_assignments, n_points * sizeof(uint32_t),          hipMemcpyDeviceToHost);

    hipFree(&d_points);
    hipFree(&d_centroids);
    hipFree(&d_assignments);
    hipFree(&d_accumulator);
    hipFree(&d_counts);
    hipFree(&d_n_points);
    hipFree(&d_n_centroids);
    hipFree(&d_n_dims);
}

CudaTimer::CudaTimer() {
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
}

void CudaTimer::start() {
    hipEventRecord(start_event, 0);
}

void CudaTimer::stop() {
    hipEventRecord(stop_event, 0);
}

float CudaTimer::elapsed_time() {
    float elapsed;
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed, start_event, stop_event);
    return elapsed;
}
